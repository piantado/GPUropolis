#include "hip/hip_runtime.h"
/*
 * GPUropolis - 2013 Aug 30 - Steve Piantadosi 
 * 
 * Main code!
 */

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <getopt.h>
#include <string.h>
#include <vector>

const float PRIOR_TEMPERATURE = 1.0; // the prior temperature
const float LL_TEMPERATURE = 1.0; // the temperature on the likelihood
float POSTERIOR_TEMPERATURE = 1.0;

const double RESAMPLE_PRIOR_TEMPERATURE = 1000.0; // when we resample, what temperatures do we use?
const double RESAMPLE_LIKELIHOOD_TEMPERATURE = 1000.0; 

#include "src/misc.cu"
#include "src/data.cu"
#include "src/hypothesis.cu"
#include "src/programs.cu"
#include "src/bayes.cu"
#include "src/virtual-machine.cu"
#include "src/kernel.cu"
#include "src/hypothesis-array.cu"

using namespace std;

int N = 1024*2;  // Hw many chains?

const int BLOCK_SIZE = 256; // WOW 16 appears to be fastest here...
int N_BLOCKS = 0; // set below

string in_file_path = "data.txt"; 
string OUT_PATH     = "run";

int SEED = -1; // Random number seed (for replicability) if -1, we use time()

int MCMC_ITERATIONS = 1000; 
int OUTER_BLOCKS = 1;
int BURN_BLOCKS = 1; // how many blocks (of MCMC_ITERATIONS each) do we burn-in?

int FIRST_HALF_DATA = 0; // use only the first half of the data
int EVEN_HALF_DATA  = 0; // use only the even half of the data

int PROPOSAL = 0x2; // a binary sum code for each type of proposal: 1: from prior, 2: standard tree-generation, 4: insert/delete moves (both)

// NO LONGER IMPLEMENTED: int PRINT_TOP = -1; // if -1 we print all, otherwise we print the top this many from each outer-loop

int MAIN_LOOP = 2; // an integer code for 
/* 
 * 1: start anew each outer loop (restart from prior)
 * 2: maintain the same chain (just print the most recent sample)
 * 3: resample via current probability given by RESAMPLE_*_TEMPERATURES
 * 4: resample from the global top (also using RESAMPLE_*_TEMPERATURES)
 */
/*
 * TODO: NOT IMPLEMENTED:
 * 5: resample from the top, penalizing by the number of samples already drawn from that hypothesis. So new things of high rank are 
// double MAIN_RESAMPLE_DISCOUNT = 1.0; // the posterior is penalized by this * [the number of chains started here], so that we will explore newer regions of the space preferentially (even if they are not high probability mass). If this is set to 0.0, then we just resample from the real posterior. If it's +inf, we only restart a chain once

// double RESAMPLE_IF_LOWER = 1000.0; // if we are this much lower than the max, we will be resampled from the top. 
*/

int NTOP = 5000; // store this many of the "top" hypotheses (for resampling). TODO: RIGHT NOW, THIS MUST BE LOWER THAN N DUE TO LACK OF CHECKS BELOW

static struct option long_options[] =
	{	
		{"in",           required_argument,    NULL, 'd'},
		{"iterations",   required_argument,    NULL, 'i'},
		{"N",            required_argument,    NULL, 'N'},
		{"out",          required_argument,    NULL, 'O'},
		{"outer",        required_argument,    NULL, 'o'},
		{"temperature",  required_argument,    NULL, 'T'},
		{"seed",         required_argument,    NULL, 's'},
		{"proposal",     required_argument,    NULL, 'p'},
		{"max-program-length",   required_argument,    NULL, 'L'},
		{"main-loop",     required_argument,        NULL, 'm'},
		{"print-top",    required_argument,    NULL, 't'},
		{"burn",         required_argument,    NULL, 'b'},
		{"first-half",   no_argument,    NULL, 'f'},
		{"even-half",    no_argument,    NULL, 'e'},
		{NULL, 0, 0, 0} // zero row for bad arguments
	};  

// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// main routine that executes on the host
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
int main(int argc, char** argv)
{	
	
	// ----------------------------------------------------------------------------
	// Parse command line
	// -----------------------------------------------------------------------
	int option_index = 0, opt=0;
	while( (opt = getopt_long( argc, argv, "bp", long_options, &option_index )) != -1 )
		switch( opt ) {
			case 'd': in_file_path = optarg; break;
			case 'i': MCMC_ITERATIONS = atoi(optarg); break;
			case 'N': N = atoi(optarg); break;
			case 'o': OUTER_BLOCKS = atoi(optarg); break;
			case 'O': OUT_PATH = optarg; break;
			case 'b': BURN_BLOCKS = atoi(optarg); break;
			case 'T': POSTERIOR_TEMPERATURE = (float)atof(optarg); break;
			case 's': SEED = (float)atof(optarg); break;
			case 'f': FIRST_HALF_DATA = 1; break;
			case 'm': MAIN_LOOP = atoi(optarg); break;
			//case 't': PRINT_TOP = atoi(optarg); break; // NO LONGER IMPLEMENTED
			case 'e': EVEN_HALF_DATA = 1; break;
			case 'p': PROPOSAL = atoi(optarg); break;
			case 'L': set_MAX_PROGRAM_LENGTH(atoi(optarg)); break;
			case 'h': // help output:
// 				cout << "Options: " << endl;
// 				cout << "\t--max-base=N         sets the maximum base to N" << endl;
// 				cout << "\t--max-power=N        sets the maximum power to N" << endl;
				return 0;
			default:
				return 1;
		}
	
	N_BLOCKS = N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1);
	
	// -----------------------------------------------------------------------
	// Set up the output files etc
	// -----------------------------------------------------------------------
	
	string SAMPLE_PATH = OUT_PATH+"/samples.txt";
	string MAP_PATH = OUT_PATH+"/MAPs.txt";
	string TOP_PATH = OUT_PATH+"/tops.txt";
// 	string SAMPLE_BINARY_PATH = OUTPATH+"/state"; // just a dump of host_hypotheses
	string LOG_PATH = OUT_PATH+"/log.txt";
	string PERFORMANCE_PATH = OUT_PATH+"/performance.txt";
	
	// -------------------------------------------------------------------------
	// Make the RNG replicable
	
	int seed;
	if(SEED==-1) {
		srand(time(NULL));
		seed = rand();
	} 
	else {
		seed = SEED;
	}

	// -------------------------------------------------------------------------
	// Write the log and performance log
	
	FILE* fp = fopen(LOG_PATH.c_str(), "w");
	if(fp==NULL) { cerr << "*** ERROR: Cannot open file:\t" << LOG_PATH <<"\n"; exit(1);}
	
	fprintf(fp, "-----------------------------------------------------------------\n");
	fprintf(fp, "-- Parameters:\n");
	fprintf(fp, "-----------------------------------------------------------------\n");
	fprintf(fp, "\tInput data path: %s\n", in_file_path.c_str());
	fprintf(fp, "\tOutput path: %s\n", OUT_PATH.c_str());
	fprintf(fp, "\tMCMC Iterations (per block): %i\n", MCMC_ITERATIONS);
	fprintf(fp, "\tBlocks: %i\n", OUTER_BLOCKS);
	fprintf(fp, "\tBurn Blocks: %i\n", BURN_BLOCKS);
	fprintf(fp, "\tN chains: %i\n", N);
	fprintf(fp, "\tTemperature: %f\n", POSTERIOR_TEMPERATURE);
	fprintf(fp, "\tSEED: %i\n", seed);
	fprintf(fp, "\tMain Loop: %i\n", MAIN_LOOP);
	fprintf(fp, "\tProposal: %i\n", PROPOSAL);
	fprintf(fp, "\tMax program length: %i\n", hMAX_PROGRAM_LENGTH);
	fprintf(fp, "\n\n");
	fclose(fp);
	
	fp = fopen(PERFORMANCE_PATH.c_str(), "w");
	if(fp==NULL) { cerr << "*** ERROR: Cannot open file:\t" << PERFORMANCE_PATH <<"\n"; exit(1);}
	fprintf(fp, "block\tdevice.time\ttransfer.time\thost.time\tsamples.per.second\tf.per.second\tprimitives.per.second\ttransfer.mb.per.second\n");
	fclose(fp);
	
	// -----------------------------------------------------------------------
	// Read the data and set up some arrays
	// -----------------------------------------------------------------------
	
	vector<datum>* data_vec = load_data_file(in_file_path.c_str(), FIRST_HALF_DATA, EVEN_HALF_DATA);
	datum* host_data = &((*data_vec)[0]); // can do this with vectors now
	
	const int DLEN = data_vec->size();
	const size_t DATA_BYTE_LEN = DLEN*sizeof(datum);
	
	// Echo the run data if we want:
// 	for(int i=0;i<DLEN;i++)	printf("# %4.4f\t%4.4f\t%4.4f\n", host_data[i].input, host_data[i].output, host_data[i].sd);
	
	// and put this on the GPU
	datum* device_data; 
	hipMalloc((void **) &device_data, DATA_BYTE_LEN);
	hipMemcpy(device_data, host_data, DATA_BYTE_LEN, hipMemcpyHostToDevice);
	
	// -----------------------------------------------------------------------
	// Set up some bits...
	// -----------------------------------------------------------------------
	
	size_t HYPOTHESIS_ARRAY_SIZE = N * sizeof(hypothesis);
	hypothesis* host_hypotheses = new hypothesis[N]; 
	hypothesis* device_hypotheses; hipMalloc((void **) &device_hypotheses, HYPOTHESIS_ARRAY_SIZE); // device allocate

	hypothesis* host_hypothesis_tmp = new hypothesis[N]; 
	
	hypothesis* host_out_MAPs = new hypothesis[N];
	hypothesis* device_out_MAPs; hipMalloc((void **) &device_out_MAPs, HYPOTHESIS_ARRAY_SIZE); // device allocate
	

	// -----------------------------------------------------------------------
	// Initialize our hypotheses
	// -----------------------------------------------------------------------
	
	// initialize this local array (copied to CUDA after any further changes)
	for(int n=0;n<N;n++){
		initialize(&host_hypotheses[n]);
		initialize(&host_out_MAPs[n]);
		initialize(&host_hypothesis_tmp[n]);
	}

	// store the top hypotheses overall
	hypothesis* host_top_hypotheses = new hypothesis[NTOP+N]; // store N so that we can push N on the end, sort, and then remove duplicates
	for(int i=0;i<NTOP+N;i++) initialize(&host_top_hypotheses[i]);
	
	// a special guy we keep  empty
	hypothesis blankhyp; 
	initialize(&blankhyp);
		
	// and copy these to device
	hipMemcpy(device_hypotheses, host_hypotheses, HYPOTHESIS_ARRAY_SIZE, hipMemcpyHostToDevice);
	
	// -----------------------------------------------------------------------
	// Main loop
	// -----------------------------------------------------------------------
	
	time_t start_t, stop_t;
	double secDEVICE, secHOST, secTRANSFER; // how long do we spend on each?
	
	for(int outer=0;outer<OUTER_BLOCKS+BURN_BLOCKS;outer++) {
		
		// increase the max program length, as a form of tempering...
		// set_MAX_PROGRAM_LENGTH(min(outer,MAX_MAX_PROGRAM_LENGTH));
		
		// -----------------------------------------------------------------------------------------------------
		// Run
		
		time(&start_t);
		kernel<<<N_BLOCKS,BLOCK_SIZE>>>(N, PROPOSAL, MCMC_ITERATIONS, POSTERIOR_TEMPERATURE, DLEN, device_data, device_hypotheses, device_out_MAPs, seed+N*outer,  (outer==0)||(MAIN_LOOP==1) );
		hipDeviceSynchronize(); // wait for preceedings requests to finish
		time(&stop_t);
		secDEVICE = difftime(stop_t, start_t);
		
		time(&start_t);
		// Retrieve result from device and store it in host array
		hipMemcpy(host_hypotheses, device_hypotheses, HYPOTHESIS_ARRAY_SIZE, hipMemcpyDeviceToHost);
		hipMemcpy(host_out_MAPs,   device_out_MAPs,   HYPOTHESIS_ARRAY_SIZE, hipMemcpyDeviceToHost);
		time(&stop_t);
		secTRANSFER = difftime(stop_t, start_t);
		
		time(&start_t);
		
		// sort them as required below
		qsort( (void*)host_out_MAPs,   N, sizeof(hypothesis), hypothesis_posterior_compare);
		qsort( (void*)host_hypotheses, N, sizeof(hypothesis), hypothesis_posterior_compare);
		
		// make sure our check bits have not changed -- that we didn't overrun anything
		for(int rank=0; rank<N; rank++){
			assert(host_hypotheses[rank].check0 == 33);
			assert(host_hypotheses[rank].check1 == 33);
			assert(host_hypotheses[rank].check2 == 33);
			assert(host_hypotheses[rank].check3 == 33);
		}

		// -----------------------------------------------------------------------------------------------------
		// Save results. But not for burn blocks
		
		// print out
		if(outer >= BURN_BLOCKS) {
			dump_to_file(SAMPLE_PATH.c_str(), host_hypotheses, N, 1); // dump samples
			dump_to_file(MAP_PATH.c_str(), host_out_MAPs, N, 1); // dump maps
		}
		
		// -----------------------------------------------------------------------------------------------------
		// Manage our collection of the top hypotheses (here: samples, not MAPs)
		// we put these in host_top_hypotheses, and the first NTOP of these always stores the best
		// we maintain this by copying all of host_out_MAPs to the end, resorting, and removing duplicates
		
		memcpy( (void*)&host_top_hypotheses[NTOP], (void*)host_out_MAPs, HYPOTHESIS_ARRAY_SIZE); // put these at the end
 		qsort(  (void*)host_top_hypotheses, NTOP+N, sizeof(hypothesis), sort_bestfirst_unique); // resort, best first, putting duplicate programs next to each other
		
		// and now delete duplicates
		for(int i=0,j=0;i<NTOP;i++,j++) {
			if(j<NTOP+N) {
				// skip forward over everything identical
				while(j+1 < NTOP+N && hypothesis_structurally_identical(&host_top_hypotheses[j], &host_top_hypotheses[j+1]))
					j++;
				
				if(j!=i) memcpy( (void*)&host_top_hypotheses[i], &host_top_hypotheses[j], sizeof(hypothesis));
			}
			else { // out of hyppotheses, so pad with blankhyp
				memcpy( (void*)&host_top_hypotheses[i], &blankhyp, sizeof(hypothesis));
				
			}
		}
		// and save this
		dump_to_file(TOP_PATH.c_str(), host_top_hypotheses, NTOP, 0);
				
		// -----------------------------------------------------------------------------------------------------
		// Now how to handle the end of a "block" -- what update do we do?
		
		if(MAIN_LOOP == 1) {
			// Then we just regenerate. This happens above by passing the last variable to device_run
		}
		else if(MAIN_LOOP == 2) {
			// Then we do nothing. Continue the same chain
		}
		else if(MAIN_LOOP == 3) {
			
			// host_hypotheses already sorted to be best-last
			
			// Resample from the current chain
			multinomial_sample(N, host_hypothesis_tmp, host_hypotheses, N, RESAMPLE_PRIOR_TEMPERATURE, RESAMPLE_LIKELIHOOD_TEMPERATURE);
			
// 			memcpy(host_hypotheses, host_hypothesis_tmp, HYPOTHESIS_ARRAY_SIZE); // probably not necessary
			// Since we modified, copy back to the device arrays
			hipMemcpy(device_hypotheses, host_hypothesis_tmp, HYPOTHESIS_ARRAY_SIZE, hipMemcpyHostToDevice);
		}
		else if(MAIN_LOOP == 4) {
		
			// resort to be best-last -- but *only* sort the top NTOP (*not* all of them!)
			qsort(  (void*)host_top_hypotheses, NTOP, sizeof(hypothesis), hypothesis_posterior_compare);
			
			// resample from the top!
			multinomial_sample(N, host_hypothesis_tmp, host_top_hypotheses, NTOP, RESAMPLE_PRIOR_TEMPERATURE, RESAMPLE_LIKELIHOOD_TEMPERATURE);
		
			// Since we modified, copy back to the device array
			hipMemcpy(device_hypotheses, host_hypothesis_tmp, HYPOTHESIS_ARRAY_SIZE, hipMemcpyHostToDevice);
		}
		time(&stop_t);
		secHOST = difftime(stop_t, start_t);
		
		// -----------------------------------------------------------------------------------------------------
		// output some performance stats
		
		double secTOTAL = secHOST + secTRANSFER + secDEVICE;
		
		unsigned long total_primitives = 0; // count up *approximately* how many primitives were evaluated
		for(int i=0;i<N;i++) total_primitives += host_hypotheses[i].program_length;
		
		FILE* fp = fopen(PERFORMANCE_PATH.c_str(), "a");
		fprintf(fp, "%i\t%.2f\t%.2f\t%.2f\t%.2f\t%.2f\t%.2f\t%.2f\n",
			 outer, 
	                 secDEVICE, 
	                 secTRANSFER, 
	                 secHOST, 
	                 double(N)*double(MCMC_ITERATIONS)*double(__builtin_popcount(PROPOSAL))/ secTOTAL,
			 double(N)*double(MCMC_ITERATIONS)*double(DLEN)*double(__builtin_popcount(PROPOSAL))/secTOTAL,
			 double(MCMC_ITERATIONS)*double(__builtin_popcount(PROPOSAL))*double(DLEN)*double(total_primitives)/secTOTAL, 
			 double(sizeof(host_out_MAPs) + sizeof(host_hypotheses))/(1048576. * secTRANSFER)   );
		fclose(fp);
	}

	// -----------------------------------------------------------------------
	// Cleanup
	// -----------------------------------------------------------------------
	
	delete[] host_hypotheses;
	delete[] host_hypothesis_tmp;
	delete[] host_data;
	
	hipFree(device_data);
	hipFree(device_hypotheses);
	hipFree(device_out_MAPs);
	
}