#include "hip/hip_runtime.h"
/*
 * GPUropolis - 2013 Aug 30 - Steve Piantadosi 
 * 
 * Main code!
 * 
 * Repetitions -- tops are maintained through reps, but samples and MAPs are distinguishable
 */

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <getopt.h>
#include <string.h>
#include <vector>

// A hacked prior to penalize extra deep X, corresponding to very weirdo functions
const float X_DEPTH_PENALTY = 100.0; // extra penalty for X depth. 0 here gives PCFG generation probability prior
const float X_PENALTY = 10.0; // penalty for using X

// Specification of the prior
// in tree resampling, the expected length here is important in getting a good acceptance rate -- too low
// (meaning too long) and we will reject almost everything
const float EXPECTED_LENGTH = 3.0; // also the expected length of proposals
const float PRIOR_XtoCONSTANT = 0.1; //what proportion of constant proposals are x (as opposed to all other constants)?

#include "src/misc.cu"
#include "src/__PRIMITIVES.cu"
#include "src/data.cu"
#include "src/hypothesis.cu"
#include "src/programs.cu"
#include "src/mcmc-specification.cu"
#include "src/mcmc-results.cu"
#include "src/virtual-machine.cu"
#include "src/hypothesis-array.cu"
#include "src/kernels/MH-simple-kernel.cu"

using namespace std;

int N = 1024;  // Hw many chains?
int NTOP = 5000; // store this many of the "top" hypotheses 

const int BLOCK_SIZE = 128; // WOW 16 appears to be fastest here...
int N_BLOCKS = 0; // set below
const int HARDARE_MAX_X_BLOCKS = 1024;
const int HARDWARE_MAX_THREADS_PER_BLOCK = 1024; // cannot exceed this many threads per block! For compute level 2.x and greater!

string in_file_path = "data.txt"; 
string OUT_PATH     = "run";

int SEED = -1; // Random number seed (for replicability) if -1, we use time()

int REPETITONS = 1; // how many outer loops do we repeat?
int MCMC_ITERATIONS = 1000; 
int OUTER_BLOCKS = 1;
int BURN_BLOCKS = 0; // how many blocks (of MCMC_ITERATIONS each) do we burn-in?

int FIRST_HALF_DATA = 0; // use only the first half of the data
int EVEN_HALF_DATA  = 0; // use only the even half of the data

int END_OF_BLOCK_ACTION = 2; // an integer code for 
/* 
 * 1: start anew each outer loop (restart from prior)
 * 2: maintain the same chain (just print the most recent sample)
 */
/*
 * TODO: NOT IMPLEMENTED:
 * 5: resample from the top, penalizing by the number of samples already drawn from that hypothesis. So new things of high rank are 
// double MAIN_RESAMPLE_DISCOUNT = 1.0; // the posterior is penalized by this * [the number of chains started here], so that we will explore newer regions of the space preferentially (even if they are not high probability mass). If this is set to 0.0, then we just resample from the real posterior. If it's +inf, we only restart a chain once

// double RESAMPLE_IF_LOWER = 1000.0; // if we are this much lower than the max, we will be resampled from the top. 
*/

static struct option long_options[] =
	{	
		{"in",           required_argument,    NULL, 'd'},
		{"iterations",   required_argument,    NULL, 'i'},
		{"repetitions",  required_argument,    NULL, 'R'},
		{"N",            required_argument,    NULL, 'N'},
		{"out",          required_argument,    NULL, 'O'},
		{"outer",        required_argument,    NULL, 'o'},
		{"temperature",  required_argument,    NULL, 'T'},
		{"seed",         required_argument,    NULL, 's'},
		{"max-program-length",   required_argument,    NULL, 'L'},
		{"end-of-block-action",     required_argument,        NULL, 'm'},
		{"print-top",    required_argument,    NULL, 't'},
		{"burn",         required_argument,    NULL, 'b'},
		{"first-half",   no_argument,    NULL, 'f'},
		{"even-half",    no_argument,    NULL, 'e'},
		{"all",    no_argument,    NULL, '_'},
		{NULL, 0, 0, 0} // zero row for bad arguments
	};  

// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// main routine that executes on the host
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
int main(int argc, char** argv)
{	
	
	// ----------------------------------------------------------------------------
	// Parse command line
	// -----------------------------------------------------------------------
	int option_index = 0, opt=0;
	while( (opt = getopt_long( argc, argv, "bp", long_options, &option_index )) != -1 )
		switch( opt ) {
			case 'd': in_file_path = optarg; break;
			case 'i': MCMC_ITERATIONS = atoi(optarg); break;
			case 'N': N = atoi(optarg); break;
			case 'o': OUTER_BLOCKS = atoi(optarg); break;
			case 'O': OUT_PATH = optarg; break;
			case 'R': REPETITONS = atoi(optarg); break;
			case 'b': BURN_BLOCKS = atoi(optarg); break;
			case 's': SEED = (float)atof(optarg); break;
			case 'f': FIRST_HALF_DATA = 1; break;
			case 'm': END_OF_BLOCK_ACTION = atoi(optarg); break;
			case 'e': EVEN_HALF_DATA = 1; break;
			case 'L': set_MAX_PROGRAM_LENGTH(atoi(optarg)); break;
			case '_': break; // don't do anything if we use all the data
			case 'h': // help output:
// 				cout << "Options: " << endl;
// 				cout << "\t--max-base=N         sets the maximum base to N" << endl;
// 				cout << "\t--max-power=N        sets the maximum power to N" << endl;
				return 0;
			default:
				return 1;
		}
	
	N_BLOCKS = N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1);
	
	assert(N_BLOCKS < HARDARE_MAX_X_BLOCKS); // can have at most this many blocks
	assert(N/N_BLOCKS <= HARDWARE_MAX_THREADS_PER_BLOCK); // MUST HAVE LESS THREADS PER BLOCK!!
	
	
	// -----------------------------------------------------------------------
	// Set up the output files etc
	// -----------------------------------------------------------------------
	
	string SAMPLE_PATH = OUT_PATH+"/samples.txt";
	string MAP_PATH = OUT_PATH+"/MAPs.txt";
	string TOP_PATH = OUT_PATH+"/tops.txt";
// 	string SAMPLE_BINARY_PATH = OUTPATH+"/state"; // just a dump of host_hypotheses
	string LOG_PATH = OUT_PATH+"/log.txt";
	string PERFORMANCE_PATH = OUT_PATH+"/performance.txt";
	
	// -------------------------------------------------------------------------
	// Make the RNG replicable
	
	int seed;
	if(SEED==-1) {
		srand(time(NULL));
		seed = rand();
	} 
	else {
		seed = SEED;
	}

	// -------------------------------------------------------------------------
	// Write the log and performance log
	
	FILE* fp = fopen(LOG_PATH.c_str(), "w");
	if(fp==NULL) { cerr << "*** ERROR: Cannot open file:\t" << LOG_PATH <<"\n"; exit(1);}
	
	fprintf(fp, "-----------------------------------------------------------------\n");
	fprintf(fp, "-- Parameters:\n");
	fprintf(fp, "-----------------------------------------------------------------\n");
	fprintf(fp, "\tInput data path: %s\n", in_file_path.c_str());
	fprintf(fp, "\tOutput path: %s\n", OUT_PATH.c_str());
	fprintf(fp, "\tRepetitions: %i\n", REPETITONS);
	fprintf(fp, "\tMCMC Iterations (per block): %i\n", MCMC_ITERATIONS);
	fprintf(fp, "\tBlocks: %i\n", OUTER_BLOCKS);
	fprintf(fp, "\tBurn Blocks: %i\n", BURN_BLOCKS);
	fprintf(fp, "\tN chains: %i\n", N);
	fprintf(fp, "\tSEED: %i\n", seed);
	fprintf(fp, "\tEnd of block action: %i\n", END_OF_BLOCK_ACTION);
	fprintf(fp, "\tMax program length: %i\n", hMAX_PROGRAM_LENGTH);
	fprintf(fp, "\tX to constant proportion: %f\n", PRIOR_XtoCONSTANT);
	
	fprintf(fp, "\n\n");
	fclose(fp);
	
	fp = fopen(PERFORMANCE_PATH.c_str(), "w");
	if(fp==NULL) { cerr << "*** ERROR: Cannot open file:\t" << PERFORMANCE_PATH <<"\n"; exit(1);}
	fprintf(fp, "repetition\tblock\tperfect.ll\tMAP.ll\tdevice.time\ttransfer.time\thost.time\tsamples.per.second\tf.per.second\tprimitives.per.second\ttransfer.mb.per.second\tacceptance.ratio\n");
	fclose(fp);
	
	// -----------------------------------------------------------------------
	// Set up the prior
	// -----------------------------------------------------------------------
	
	assert(NUM_OPS < MAX_NUM_OPS);  // check that we don't have too many for our array

	// how many times have we seen each number of args?
	int count_args[] = {0,0,0};
	for(int i=1;i<NUM_OPS;i++){  // skip NOOP
		assert( hNARGS[i] <= 2); // we must have this to compute expected lengths correctly. It can be changed for arbitrary-arity later
		count_args[ hNARGS[i]]++; 
	}
		
	/*
	* The expected length satisfies:
	* E = p0arg + p1arg(E+1) + p2arg(2 E + 1)
	* E = p0arg+p1arg+p2arg + E ( p1arg + 2 p2arg)
	* E = 1 + E (p1arg + 2 p2arg)
	* 1 = E (1-p1arg-p2arg)
	* so
	* E = 1/(1-p1arg - 2 p2arg)
	* 
	* Constraining p1arg = p2arg,
	* E = 1/(1-3p1arg)
	* 
	* so
	* 
	* p1arg = p2arg = (1-1/E)/3
	* and then we must account for the number in each class
	*/
	float P = (1.0-1.0/EXPECTED_LENGTH)/3.0;
	float P_0arg = (1.0-2.0*P);
	float P_X        = P_0arg * PRIOR_XtoCONSTANT;
	float P_CONSTANT = P_0arg * (1.0-PRIOR_XtoCONSTANT) / float(count_args[0]-1);
	// This way will divide evenly between 1- and 2- args
// 	float P_1arg  = P / float(count_args[1]);
// 	float P_2arg  = P / float(count_args[2]);
	// This way will put all mass equally among all functions, regardless of arity:
	float P_1arg  = P / float(count_args[1] + count_args[2]);
	float P_2arg  = P / float(count_args[1] + count_args[2]);;
	
	for(int i=0;i<MAX_NUM_OPS;i++) hPRIOR[i] = 0.0; // must initialize since not all will be used
	
	for(int i=0;i<NUM_OPS;i++) {
		if( i == NOOP_ )         { hPRIOR[i] = 0.0; }
		else if( i == X_ )       { hPRIOR[i] = P_X; }
		else if( hNARGS[i] == 0) { hPRIOR[i] = P_CONSTANT; }
		else if( hNARGS[i] == 1) { hPRIOR[i] = P_1arg; }
		else if( hNARGS[i] == 2) { hPRIOR[i] = P_2arg; }	
	}

	// normalize the prior
	double priorZ = 0.0;
	for(int i=0;i<NUM_OPS;i++) priorZ += hPRIOR[i];
	for(int i=0;i<NUM_OPS;i++) hPRIOR[i] /= priorZ;

	// and copy PRIOR over to the device
	hipMemcpyToSymbol(HIP_SYMBOL(dPRIOR), hPRIOR, MAX_NUM_OPS*sizeof(float), 0, hipMemcpyHostToDevice);
	
	// Echo the prior
	fp = fopen(LOG_PATH.c_str(), "a");
	if(fp==NULL) { cerr << "*** ERROR: Cannot open file:\t" << LOG_PATH <<"\n"; exit(1);}
	
	fprintf(fp, "\n-----------------------------------------------------------------\n");
	fprintf(fp, "-- Prior:\n");
	fprintf(fp, "-----------------------------------------------------------------\n");
	for(int i=0;i<NUM_OPS;i++) 
		fprintf(fp, "\t%i\t%s\t%f\n", i, NAMES[i], hPRIOR[i]);
	fclose(fp);

	// -----------------------------------------------------------------------
	// Read the data and set up some arrays
	// -----------------------------------------------------------------------
	
	vector<datum>* data_vec = load_data_file(in_file_path.c_str(), FIRST_HALF_DATA, EVEN_HALF_DATA);
	datum* host_data = &((*data_vec)[0]); // can do this with vectors now
	
	const int DLEN = data_vec->size();
	const size_t DATA_BYTE_LEN = DLEN*sizeof(datum);

	// compute the maximum possible ll
	// we use this for the start of annealing temperature
	double PERFECT_LL = 0.0;
	for(int di=0;di<DLEN;di++) {PERFECT_LL += lnormalpdf( 0.0, host_data[di].sd); }

	// and put this on the GPU
	datum* device_data; 
	hipMalloc((void **) &device_data, DATA_BYTE_LEN);
	hipMemcpy(device_data, host_data, DATA_BYTE_LEN, hipMemcpyHostToDevice);
	
	// Echo the data we actually run with (post-filtering for even/firsthalf)
	fp = fopen(LOG_PATH.c_str(), "a");
	fprintf(fp, "\n-----------------------------------------------------------------\n");
	fprintf(fp, "-- Data:\n");
	fprintf(fp, "-----------------------------------------------------------------\n");
	for(int i=0;i<DLEN;i++) 
		fprintf(fp, "\t%f\t%f\t%f\n", host_data[i].input, host_data[i].output, host_data[i].sd);
	fclose(fp);
	
	// -----------------------------------------------------------------------
	// Initialize RNG
	// -----------------------------------------------------------------------
	
	// define random numbers:
	int rx,ry,rz,rw;
	rx = rand(); ry = rand(); rz = rand(); rw = rand();
	
	// this is how we seed each chain
	long rng_seed = 0;
	
	// -----------------------------------------------------------------------
	// Set up some bits...
	// -----------------------------------------------------------------------
	
	// For locally manipulating hypotheses
	size_t HYPOTHESIS_ARRAY_SIZE = N * sizeof(hypothesis);
	hypothesis* host_hypotheses = new hypothesis[N]; 
	hypothesis* host_hypothesis_tmp = new hypothesis[N]; 
	hypothesis* host_out_MAPs = new hypothesis[N];
	
	int MCMC_RESULTS_SIZE = sizeof(mcmc_results)*N;
	mcmc_results* host_mcmc_results = new mcmc_results[N];
	mcmc_results* device_mcmc_results; hipMalloc((void **) &device_mcmc_results, MCMC_RESULTS_SIZE); // device allocate
	
	// initialize this local array (copied to CUDA after any further changes)
	for(int n=0;n<N;n++){
		initialize(&(host_mcmc_results[n].MAP),    RNG_ARGS);
		initialize(&(host_mcmc_results[n].sample), RNG_ARGS);
	}
	
	// store the top hypotheses overall
	hypothesis* host_top_hypotheses = new hypothesis[NTOP+2*N]; 
	for(int i=0;i<NTOP+2*N;i++) initialize(&host_top_hypotheses[i], RNG_ARGS);
	
	// a special guy we keep  empty
	hypothesis blankhyp; initialize(&blankhyp, RNG_ARGS);
	
	// -----------------------------------------------------------------------
	// Make the specification for mcmc steps, iterations, etc.
	// -----------------------------------------------------------------------
	
	int NSPECSIZE = N*sizeof(mcmc_specification);
	
	mcmc_specification* host_spec = new mcmc_specification[N]; 
	mcmc_specification* dev_spec; hipMalloc((void **) &dev_spec, NSPECSIZE ); // device allocate

	
	// And set up the specifications!
	for(int i=0;i<N;i++) {
		host_spec[i].prior_temperature = 1.0;
		host_spec[i].likelihood_temperature = 1.0;
		host_spec[i].acceptance_temperature = 1.0;
		host_spec[i].iterations = MCMC_ITERATIONS; // how many steps to run?
		host_spec[i].initialize = 1;
		host_spec[i].rng_seed = 0; // set below
		host_spec[i].data_length = DLEN;
		host_spec[i].data = device_data;
		// TODO: Max program length should be in here! But that's hard, unless that's also a feature of hyp...
	}
	
	
	// -----------------------------------------------------------------------
	// Main loop
	// -----------------------------------------------------------------------
	
	clock_t mytimer;
	
	
	for(int rep=0; rep<REPETITONS; rep++) { // how many repetitions do we do?
	for(int outer=0;outer<OUTER_BLOCKS+BURN_BLOCKS;outer++) {
		double secDEVICE=0.0, secHOST=0.0, secTRANSFER=0.0; // how long do we spend on each?	
		
		// increase the max program length, as a form of tempering...
		// set_MAX_PROGRAM_LENGTH(min(outer,MAX_MAX_PROGRAM_LENGTH));
		
		// Set the specifications
		for(int i=0;i<N;i++) {
			host_spec[i].initialize = (outer==0)||(END_OF_BLOCK_ACTION==1);
			
			
			host_spec[i].rng_seed = seed + rng_seed++; // set this seed
// 			cerr << host_spec[i].initialize << "\t" << host_spec[i].rng_seed << endl;
			// THIS IS ANNEALING ON THE LIKELIHOOD
// 			host_spec[i].likelihood_temperature = 1.0 + 5.0 / float(outer+1.0);
		}
		mytimer = clock();
		hipMemcpy(dev_spec, host_spec, NSPECSIZE, hipMemcpyHostToDevice);
		hipMemcpy(device_mcmc_results, host_mcmc_results, MCMC_RESULTS_SIZE, hipMemcpyHostToDevice);
		secTRANSFER += double(clock() - mytimer) / CLOCKS_PER_SEC;
		
		//////////////// Now run: //////////////// 
		
		mytimer = clock();
		
		hipDeviceSynchronize(); 
		MH_simple_kernel<<<N_BLOCKS,BLOCK_SIZE>>>(N, dev_spec, device_mcmc_results);
		hipDeviceSynchronize(); // wait for preceedings requests to finish
		
		secDEVICE = double(clock() - mytimer) / CLOCKS_PER_SEC;
				
		// Retrieve result from device and store it in host array
		mytimer = clock();
		hipMemcpy(host_mcmc_results, device_mcmc_results, MCMC_RESULTS_SIZE, hipMemcpyDeviceToHost);
		secTRANSFER += double(clock() - mytimer) / CLOCKS_PER_SEC;
		
		mytimer = clock(); // for timing the rest of host operations
		
		// and copy the host hypotheses over to hosts and maps
		for(int i=0;i<N;i++){ 
// 			cerr <<  i << "\t" << host_mcmc_results[i].sample.posterior << endl;
			COPY_HYPOTHESIS( &host_hypotheses[i], &(host_mcmc_results[i].sample));
			COPY_HYPOTHESIS( &host_out_MAPs[i],   &(host_mcmc_results[i].MAP));
		}		
		
		// sort them as required below
		qsort( (void*)host_out_MAPs,   N, sizeof(hypothesis), hypothesis_posterior_compare);
		qsort( (void*)host_hypotheses, N, sizeof(hypothesis), hypothesis_posterior_compare);
		
		// make sure our check bits have not changed -- that we didn't overrun anything
		for(int rank=0; rank<N; rank++){
			assert(host_hypotheses[rank].check0 == CHECK_BIT);
			assert(host_hypotheses[rank].check1 == CHECK_BIT);
			assert(host_hypotheses[rank].check2 == CHECK_BIT);
			assert(host_hypotheses[rank].check3 == CHECK_BIT);
			assert(host_hypotheses[rank].check4 == CHECK_BIT);
			assert(host_hypotheses[rank].check5 == CHECK_BIT);
			assert(host_hypotheses[rank].check6 == CHECK_BIT);
		}

		// -----------------------------------------------------------------------------------------------------
		// Save results. But not for burn blocks
		
		// print out
		if(outer >= BURN_BLOCKS) {
			dump_to_file(SAMPLE_PATH.c_str(), host_hypotheses, rep, outer, N, 1); // dump samples
			dump_to_file(MAP_PATH.c_str(), host_out_MAPs, rep, outer, N, 1); // dump maps
		}
		
		// -----------------------------------------------------------------------------------------------------
		// Manage our collection of the top hypotheses (here: samples, not MAPs)
		// we put these in host_top_hypotheses, and the first NTOP of these always stores the best
		// we maintain this by copying all of host_out_MAPs to the end, resorting, and removing duplicates
		
		memcpy( (void*)&host_top_hypotheses[NTOP], (void*)host_out_MAPs, HYPOTHESIS_ARRAY_SIZE); // put these at the end
		memcpy( (void*)&host_top_hypotheses[NTOP+N], (void*)host_hypotheses, HYPOTHESIS_ARRAY_SIZE); // put these at the end
		
		
		// resort, best first, putting duplicate programs next to each other
 		qsort(  (void*)host_top_hypotheses, NTOP+2*N, sizeof(hypothesis), sort_bestfirst_unique); 
		// and now delete duplicates
		delete_duplicates(host_top_hypotheses, NTOP, NTOP+2*N, &blankhyp);
		// and save this
		dump_to_file(TOP_PATH.c_str(), host_top_hypotheses, rep, outer, NTOP, 0);
				
		// -----------------------------------------------------------------------------------------------------
		// Now how to handle the end of a "block" -- what update do we do?
			
		secHOST = double(clock() - mytimer) / CLOCKS_PER_SEC;
		
		// -----------------------------------------------------------------------------------------------------
		// output some performance stats (we do this for burn blocks)
		
		double secTOTAL = secHOST + secTRANSFER + secDEVICE;
		
		unsigned long total_primitives = 0; // count up *approximately* how many primitives were evaluated
		for(int i=0;i<N;i++) total_primitives += host_hypotheses[i].program_length;
		
		double AR_mean=0.0; // get the accept/reject mean
		for(int i=0;i<N;i++) 
			AR_mean += float(host_mcmc_results[i].acceptance_count) / float(host_mcmc_results[i].proposal_count);
		AR_mean /= float(N);
		
		FILE* fp = fopen(PERFORMANCE_PATH.c_str(), "a");
		fprintf(fp, "%i\t%i\t%.2f\t%.2f\t%.6f\t%.6f\t%.6f\t%.2f\t%.2f\t%.2f\t%.2f\t%.5f\n",
			rep, 
			outer, 
			PERFECT_LL,
			host_top_hypotheses[0].likelihood, 
			secDEVICE, 
			secTRANSFER, 
			secHOST, 
			double(N)*double(MCMC_ITERATIONS)/ secTOTAL,
			double(N)*double(MCMC_ITERATIONS)*double(DLEN)/secTOTAL,
			double(MCMC_ITERATIONS)*double(DLEN)*double(total_primitives)/secTOTAL, 
			double(HYPOTHESIS_ARRAY_SIZE*2)/(1048576. * secTRANSFER),
			AR_mean
       		);
		fclose(fp);
	
	} // end outer loop 
	} // end repetitions
	
	// -----------------------------------------------------------------------
	// Cleanup
	// -----------------------------------------------------------------------
	
	delete[] host_hypotheses;
	delete[] host_hypothesis_tmp;
	delete[] host_data;
	delete[] host_mcmc_results;
	delete[] host_spec;
	
	hipFree(device_data);
	hipFree(dev_spec);
	hipFree(device_mcmc_results);
	
}