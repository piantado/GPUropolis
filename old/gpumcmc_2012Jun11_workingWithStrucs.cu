#include "hip/hip_runtime.h"
/*
 * -- To run on CPU: http://code.google.com/p/gpuocelot/
 * -- If we want the prior not to depend on the scale, we must build in some logarithmic steps, and treat them equally
 * -- If we want the likelihood not to depend on the scale, we may want a ratio likelihood... but the downside of that is that its not shift-invariant (so if you shift near zero, things go to hell)
 * - Once you add samples, you can resample for a number of steps and output every number of steps
 * - REPLACE &(x[0]) with &x, I think will work!
 * 
 * TODO:
 * 	- Bundle together a hypothesis so its easy to use, and you can get rid of the posterior return
 * 	- Create a local copy of the input and output arrays
 * 	- Can run at a variety of temperatures
 * 	- We can examine the "health" of the chain by seeing how many times we find the "top" one
 * 	-- Huh, probably by particle re-sampling, we can explore the really good regions of the space even better since that moves everyone to one of those regions
 */

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>

#include <vector>
using namespace std;

// this is our arithmetic if/then macro
#define ifthen(x,y,z) ((y)*(x) + (z)*(1-(x)))

// Check if something is not a nan or inf
#define is_valid(x) (!(isnan(x) || isinf(x)))

const int N = 1000;  // oHw many chains?
const int BLOCK_SIZE = 512;
const int N_BLOCKS = N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1);
	
// int DLEN; // how long is the data?
// float* INPUT; // = { 1.0, 2.0, 3.0, 4.0,5.0, 6.0, 7.0 };
// float* OUTPUT;// = { 1.0000000, 0.7071068, 0.5773503, 0.5000000, 0.4472136, 0.4082483, 0.3779645}; // x**-0.5
// // __constant__ const float OUTPUT[DLEN] = { 1.0, 4.0, 9.0, 16.0, 25.0, 36.0, 49.0 };
// __constant__ const float OUTPUT[DLEN] = { 0.000000000, -0.6065307, -1.4330626, -2.3364023, -3.2749230, -4.2324086, -5.2012674 }; // (1-x) / exp(1/x)

// The file that we read input from
char* INPUT_FILE = "datasets/squared.txt"; //"datasets/squared.txt";
// __constant__ const float OUTPUT[DLEN] = { 0.0, 0.4041569,0.9990636,1.5557204,2.0408044,2.4605594,2.8264565}; // log(1+x*(x-1)) / (exp(2/x))

const float LL_SIGMA     = 0.10; //0.01;
const float PRIOR_TEMPERATURE = 0.1000;
const int MCMC_ITERATIONS = 1000;

const float RESAMPLE_P = 0.15; // when we propose, with what probability do we change each program element?

const int MAX_PROGRAM_LENGTH = 20; 

// CONSTANTS, 2-ARY, 1-ARY
// NUM_OPS stores the number of opsn
// TODO: IF YOU CHANGE THESE, MYOU MSUT CHANGE THE CONDITION BELOW FOR DECODING WHAT HAPPENS TO THE TOP OF STACK
enum OPS                              { ZERO, ONE, X, PI, E, ADD, SUB, MUL, DIV, POW,     LOG, EXP, NUM_OPS };
__constant__ int NARGS[NUM_OPS]     = {   0,   0,  0,  0, 0,  2,   2,   2,   2,   2,        1,  1 }; // how many args for each op?
__constant__ float NUM_OPSf = float(NUM_OPS);

enum HELD_OUT { NONE, HALF, ODD}; // which data elements do we hold out?

// how much does each op change the stack?
#define stack_change(x) (1-NARGS[x]) 

// A struct for storing a hypothesis, which is a program and some accoutrements
typedef struct {
	float prior;
	float likelihood;
	float posterior;
	float temperature;
	int program[MAX_PROGRAM_LENGTH];
} hypothesis;


// swap two pointers conditionally, without branching
__device__ void ifthenswap(int Q, void** x, void** y, void** tmp) {
	int i = int(Q>0);
	tmp[0] = *x;
	tmp[1] = *y;
	void* t1 = tmp[i];
	void* t2 = tmp[1-i];
	(*x) = t1;
	(*y) = t2;
}

// a random number 0..(n-1), using the stored locations for x,y,z,q
__device__ int random_int(int n, int& x, int& y, int& z, int& w) {
	int t;
 
	t = x ^ ( x << 11);
	x = y; y = z; z = w;
	w = w ^ (w >> 19) ^ (t ^ (t >> 8));
	
	return (w%n);
}

__device__ float random_float(int& x, int& y, int& z, int& w) {
	return float(random_int(1000000, x,y,z,w)) / 1000000.0;
}

__device__ float f_output(float x, int* program, float* registers, float* stack) {

	// first zero the stack
	for(int i=0;i<2*MAX_PROGRAM_LENGTH;i++) stack[i] = 0.0; 
	
	registers[ZERO] = 0.0f;
	registers[ONE]  = 1.0f;
	registers[X]    = x;
	registers[PI]   = 3.14159;
	registers[E]    = 2.71828;
	
	int top = MAX_PROGRAM_LENGTH; //  We start in the middle of the stack
	for(int p=0;p<MAX_PROGRAM_LENGTH;p++) { // program pointer
		int op = program[p];
		
		// update the virtual registers
		registers[ADD]  = stack[top] + stack[top-1];
		registers[SUB]  = stack[top] - stack[top-1];
		registers[MUL]  = stack[top] * stack[top-1];
		registers[DIV]  = stack[top] / stack[top-1];
		registers[POW]  = pow(stack[top], stack[top-1]);
		registers[LOG]  = log(stack[top]);
		registers[EXP]  = exp(stack[top]);
		
		// the *(op<NUM_OPS&&op>=0) here makes us handle accidentally huge numbers, treating them as 0s
		// what elements changes? If we have a constant, we push; a 1-ary and we replace; a 2-ary and we gobble one
		top += stack_change(op)*(op<NUM_OPS&&op>=0); // so 0 args push, 1 args make no change, 2args eat the top
		stack[top] = registers[op*(op<NUM_OPS&&op>=0)];
	}
// 	
	return stack[top];
}

__device__ float compute_likelihood(int DLEN, float* device_INPUT, float* device_OUTPUT, int* program, float* registers, float* stack) {

	float ll = 0.0;
	for(int i=0;i<DLEN;i++){
		// compute the difference between the output and what we see
		float d = device_OUTPUT[i] - f_output( device_INPUT[i], program, registers, stack);
		
		// and use a gaussian likelihood
		ll += (-d*d)/(2.0*LL_SIGMA*LL_SIGMA) - 0.5 * log(2.0*3.141592653589*LL_SIGMA*LL_SIGMA);
	}
	return ll;
}

// Compute the program's prior, 2^-length, but note that the program must be pushed to the "right" in the array,
// so we can do this by counting the number of leading zeros (TODO: WHICH ISN'T QUITE RIGHT)
__device__ float compute_prior(int* program) {
	int keep = 1;
	int counter = 0;
	for(int i=0;i<MAX_PROGRAM_LENGTH;i++) {
		keep = (keep && program[i] == ZERO);
		counter += keep;
	}
	return float(-(MAX_PROGRAM_LENGTH-counter))/float(PRIOR_TEMPERATURE);
}

__device__ void compute_posterior(int DLEN, float* device_INPUT, float* device_OUTPUT, hypothesis* h, float* registers, float* stack) {
	h->prior      = compute_prior(h->program);
	h->likelihood = compute_likelihood(DLEN, device_INPUT, device_OUTPUT, h->program, registers, stack);
	h->posterior = h->prior + h->likelihood;
}

// Puts a random closed expression into buf, pushed to the rhs
// and returns the start position of the expression. 
// NOTE: Due to size constraints, this may fail, which is okay, it will 
// just create a garbage proposal
__device__ int random_closed_expression(int* buf, int& x, int& y, int& z, int& w) {
	
	int nopen = -1; // to begin, we are looking for 1 arg to the left
	int notdone = 1;
	int len = 0;
	for(int i=MAX_PROGRAM_LENGTH-1;i>=0;i--) {
		int newop = random_int(NUM_OPS,x,y,z,w);
		buf[i] = notdone * newop;
		
		nopen += notdone * stack_change(newop); //(1-NARGS[newop]);
		len += notdone;

		notdone = notdone && (nopen != 0); // so when set done=0 when nopen=0 -- when we've reached the bottom!
	}
	
	return (MAX_PROGRAM_LENGTH-len);	
}

// Starting at i in buf, go forwards and return the position of 
// the matching paren (corresponding to an expression)
__device__ int find_close_backwards(int* buf, int pos) {
	
	int nopen = -1;
	int notdone = 0;
	int ret = pos+1;
	for(int i=MAX_PROGRAM_LENGTH-1;i>=0;i--) {
		notdone =  (i==pos) || (notdone && nopen != 0);
		nopen  +=  (i<=pos && notdone) * stack_change(buf[i]);
		ret    -=  (i<=pos && notdone); 
	}
	return ret;	
}

// set ar[start_ar...(start_ar+len_ar)] = x[MAX_PROGRAM_LENGTH-len_x:MAX_PROGRAM_LENGTH-1];
// nonincludsive of end_ar, but inclusive of start_ar. inclusive of start_x
// NOTE: WE can get garbage on the left of our string if we insert soemthing in 0th (rightmost) position
__device__ void special_splice(int* ar, int start_ar, int end_ar, int* x, int start_x, int* dest) {
	
	// correct for the mis-alignment of x and the gap of what its replacing
	int shift = (MAX_PROGRAM_LENGTH - start_x) - (end_ar+1-start_ar); 
	
	int xi = start_x;
	for(int ari=0;ari<MAX_PROGRAM_LENGTH;ari++) {
		int in_splice_region = (ari>=start_ar-shift) && (ari<=end_ar);
		int in_final_region = (ari > end_ar);
		
		// wrap this for in case it goes off the end
		int ar_ari_shift = ifthen( (ari+shift < 0) || (ari+shift >= MAX_PROGRAM_LENGTH), 0, ar[ari+shift]); 
// 		dest[ari] = ifthen(in_splice_region, 2, ifthen(in_final_region, 3, 1) );
		
		dest[ari] = ifthen(in_splice_region, x[xi], ifthen(in_final_region, ar[ari], ar_ari_shift) );
		
		xi += in_splice_region; // when in the splice region, increment by 1
	}
	
}

// starting at the RHS, go until we find the "Effective" program length (ignoring things that are not used)
__device__ int find_program_length(int* program) {
	return MAX_PROGRAM_LENGTH-find_close_backwards(program, MAX_PROGRAM_LENGTH-1);
}

// Kernel that executes on the CUDA device
__global__ void device_run(int N, int DLEN, float* device_INPUT, float* device_OUTPUT, hypothesis* out_hypotheses, int seed)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Set up for the RNG
	int rx = idx*seed; // 123456789; // MUST be 32 bit //!!                    <- TODO: HERE WE UNDID THE RANDOM SEED
	int ry = 362436069;
	int rz = 521288629;
	int rw = 88675123;
	
	float stack[2*MAX_PROGRAM_LENGTH]; // Stack is twice as big as program, and populated with 0s so we can eval, e.g., a whole prog of ADDs
	float registers[NUM_OPS];
	
	hypothesis current_; hypothesis* current = &current_;
	hypothesis proposal_; hypothesis* proposal = &proposal_;
	
	int program_buf_[MAX_PROGRAM_LENGTH]; // needed below as a buffer program
	int* program_buf = &(program_buf_[0]);
	
	void* swaptmp[2]; // used for swapping without arithmetic
		
	// randomly initialize
	random_closed_expression(current->program,  rx,ry,rz,rw);

	compute_posterior(DLEN, device_INPUT, device_OUTPUT, current, &(registers[0]), &(stack[0]));

	for(int mcmci=0;mcmci<MCMC_ITERATIONS;mcmci++) {
		
		
		for(int i=0;i<2;i++) { // for each kind of proposal

			float fb = 0.0;  // the forward-backward probability
			int len = find_program_length(current->program); // TODO: CHECK OFF BY !
			
			if(i==0) { // a normal tree regeneration proposal
				
				// first find a proposal location:
				int end_ar = (MAX_PROGRAM_LENGTH-1) - random_int(len-1, rx,ry,rz,rw); // the position of what we replace
				int start_ar = find_close_backwards(current->program, end_ar);
				
				// generate a novel tree in program_buf
				int start_x = random_closed_expression(program_buf, rx,ry,rz,rw);
				special_splice(current->program, start_ar, end_ar, program_buf, start_x, proposal->program); // And insert this where we wanted:
				
				float proposed_len = (float) find_program_length(proposal->program);
				float current_len = (float) find_program_length(current->program);
				
				// forward-back probability
				fb = (-log(current_len)-proposed_len*log(NUM_OPSf)) - (-log(proposed_len)-current_len*log(NUM_OPSf));
			}
			else if(i==1) { // full regeneration proposal
				
				random_closed_expression(proposal->program, rx, ry, rz, rw);
				fb = 0.0; // for using just generate random proposals
			}
			else if(i==2) {
				
				// Simple stupid proposal that just mutate at random:	 
				for(int i=0;i<MAX_PROGRAM_LENGTH;i++) {
					int r = random_float(rx,ry,rz,rw) < RESAMPLE_P;
					proposal->program[i] = ifthen(r, random_int(NUM_OPS,rx,ry,rz,rw), current->program[i]);
				}
				//fb = 0.0;
			}
			else if(i==3) {
				
				/// TODO: ON THESE FIX F/B
				
				int pos = (MAX_PROGRAM_LENGTH-1) - random_int(len-1, rx,ry,rz,rw);
				
				// random insert into a program
				for(int i=0;i<MAX_PROGRAM_LENGTH;i++) {
					proposal->program[i] = current->program[i+1]*(i<pos) + random_int(NUM_OPS,rx,ry,rz,rw)*(i==pos) + current->program[i]*(i>pos);
				}
				
				fb = (-log(float(len)) - log(NUM_OPSf) ) - (-log(float(len+1)) );
				
			}
			else if(i==4) {
				int pos = (MAX_PROGRAM_LENGTH-1) - random_int(len-1, rx,ry,rz,rw);
				
				// random delete from a program
				proposal->program[0] = 0x0;				
				for(int i=1;i<MAX_PROGRAM_LENGTH;i++) {
					proposal->program[i] = current->program[i-1]*(i<=pos) + current->program[i]*(i>pos);
				}
				
				fb = (-log(float(len)) ) - (-log(float(len-1)) - log(NUM_OPSf) );
				
			}
			
			// compute the posterior for the proposal
			compute_posterior(DLEN, device_INPUT, device_OUTPUT, proposal, &(registers[0]), &(stack[0]));
			
			// compute whether not we accept the proposal, while rejecting infs and nans
			int swap = (random_float(rx,ry,rz,rw) < exp(proposal->posterior - current->posterior - fb) && is_valid(proposal->posterior)) || (! is_valid(current->posterior));
			
			// Use a trick to swap pointers
			ifthenswap(swap, (void**)&current, (void**)&proposal, swaptmp);
			
		} // end for each proposal kind
	}
	
	if (idx<N) {
		memcpy(out_hypotheses + idx, (void*)current, sizeof(hypothesis));
	}
}
 

//------------
// For easier displays
const int MAX_OP_LENGTH = 256; // how much does each string add at most?
char SS[MAX_PROGRAM_LENGTH*2][MAX_OP_LENGTH*MAX_PROGRAM_LENGTH]; 
void print_program_as_expression(int* program) {
	
	char buf[MAX_PROGRAM_LENGTH*MAX_PROGRAM_LENGTH];
	
	int top = MAX_PROGRAM_LENGTH; // top of the stack
	
	// re-initialize our buffer
	for(int r=0;r<MAX_PROGRAM_LENGTH*2;r++) strcpy(SS[r], "0"); // since everything initializes to 0
	
	for(int p=0;p<MAX_PROGRAM_LENGTH;p++) {
		int op = program[p];
		
		switch(op) {
			case ZERO: 
				top += 1;
				strcpy(SS[top], "0");
				break;
			case ONE: 
				top += 1;
				strcpy(SS[top], "1");
				break;
			case X:
				top += 1;
				strcpy(SS[top], "x");
				break;
			case PI:
				top += 1;
				strcpy(SS[top], "PI");
				break;
			case E:
				top += 1;
				strcpy(SS[top], "E");
				break;
			case ADD:
				strcpy(buf, "(+ ");
				strcat(buf, SS[top]);
				strcat(buf, " ");
				strcat(buf, SS[top-1]);
				strcat(buf, ")");
				top -= 1;
				strcpy(SS[top], buf);
				break;
				
			case SUB:
				strcpy(buf, "(- ");
				strcat(buf, SS[top]);
				strcat(buf, " ");
				strcat(buf, SS[top-1]);
				strcat(buf, ")");
				top -= 1;
				strcpy(SS[top], buf);
				break;
				
			case MUL:
				strcpy(buf, "(* ");
				strcat(buf, SS[top]);
				strcat(buf, " ");
				strcat(buf, SS[top-1]);
				strcat(buf, ")");
				top -= 1;
				strcpy(SS[top], buf);
				break;
				
			case DIV:
				strcpy(buf, "(/ ");
				strcat(buf, SS[top]);
				strcat(buf, " ");
				strcat(buf, SS[top-1]);
				strcat(buf, ")");
				top -= 1;
				strcpy(SS[top], buf);
				break;
			case POW:
				strcpy(buf, "(pow ");
				strcat(buf, SS[top]);
				strcat(buf, " ");
				strcat(buf, SS[top-1]);
				strcat(buf, ")");
				top -= 1;
				strcpy(SS[top], buf);
				break;	
				
			case LOG:
				strcpy(buf, "(log ");
				strcat(buf, SS[top]);
				strcat(buf, ")");
				strcpy(SS[top], buf);
				break;
				
			case EXP:
				strcpy(buf, "(exp ");
				strcat(buf, SS[top]);
				strcat(buf, ")");
				strcpy(SS[top], buf);
				break;
				
		}
	}
	
	printf("%s", SS[top]);
}
 
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// main routine that executes on the host
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
int main(void)
{	
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	// The data:
	FILE* fp = fopen(INPUT_FILE, "r");
	if(fp==NULL) { cerr << "*** ERROR: Cannot open file:\t" << INPUT_FILE <<"\n"; return 1;}
	
	vector<float> inx; float x;
	vector<float> iny; float y;
	char* line = NULL; size_t len=0;
	while( getline(&line, &len, fp) != -1) {
		if (sscanf(line, "%f\t%f\n", &x, &y) == 2) { 
			inx.push_back(x);
			iny.push_back(y);
		}
	}
	
	assert(inx.size() == iny.size());
	const int DLEN = inx.size();
	const size_t DATA_BYTE_LEN = DLEN*sizeof(float);
	
	size_t out_hypotheses_size = N * sizeof(hypothesis);
	hypothesis* host_out_hypotheses = new hypothesis[N]; 
	hypothesis* device_out_hypotheses; hipMalloc((void **) &device_out_hypotheses, out_hypotheses_size); // device allocate
	 
	
	// copy the read input to the GPU
	float* device_INPUT;
	hipMalloc((void **) &device_INPUT, DATA_BYTE_LEN);
	float* host_INPUT = (float*) malloc(DATA_BYTE_LEN);
	for(int i=0;i<DLEN;i++) host_INPUT[i] = inx[i];
	hipMemcpy(device_INPUT, host_INPUT, DATA_BYTE_LEN, hipMemcpyHostToDevice);
	
	float* device_OUTPUT;
	hipMalloc((void **) &device_OUTPUT, DATA_BYTE_LEN);
	float* host_OUTPUT = (float*) malloc(DATA_BYTE_LEN);
	for(int i=0;i<DLEN;i++) host_OUTPUT[i] = iny[i];
	hipMemcpy(device_OUTPUT, host_OUTPUT, DATA_BYTE_LEN, hipMemcpyHostToDevice);
	
	// Do calculation on device:
	srand(time(NULL));
	int seed = rand();
	printf("# Starting running\n");
	device_run<<<N_BLOCKS,BLOCK_SIZE>>>(N, DLEN, device_INPUT, device_OUTPUT, device_out_hypotheses, seed);
	printf("# Done running\n");
	
	// Retrieve result from device and store it in host array
	hipMemcpy(host_out_hypotheses, device_out_hypotheses, out_hypotheses_size, hipMemcpyDeviceToHost);
	
	//Print results
	for(int n=0; n<N; n++){
		printf("%d\t%4.2f\t%4.2f\t%4.2f\t\"", n, host_out_hypotheses[n].prior+host_out_hypotheses[n].likelihood,  host_out_hypotheses[n].prior, host_out_hypotheses[n].likelihood);
		
		for(int i=0;i<MAX_PROGRAM_LENGTH;i++) printf("%d ", host_out_hypotheses[n].program[i]);
		
		printf("\"\t\"");
		
		print_program_as_expression( host_out_hypotheses[n].program );
		
		printf("\"\n");
	}
	
// 	// Cleanup
// 	delete[] host_out_hypotheses;
// 	delete[] host_OUTPUT;
// 	delete[] host_INPUT;
// 	
// 	hipFree(device_OUTPUT);
// 	hipFree(device_INPUT);
// 	hipFree(device_out_hypotheses);
}