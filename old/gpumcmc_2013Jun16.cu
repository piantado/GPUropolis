#include "hip/hip_runtime.h"
/*
 * 
 * 
 * 
 * 
 *  - DEfine a "program type" (currently int) for programs, and then we can test out short and other things
 * 
 * 
 * 
 * 
 * 
 * 
 * 
 * 
 * sORT IS NOT WORKING RIGHT
 * FOr SOME REASON WE GET THESE CRAZY HIGH PRIOR+LIKELIHOODS AND HTE SUMS ARE NONSESE
 * 
 * ANd if w efix the prior or likleihod, we get garbage, with all 0s in the proposal etc. 
 * 
 * 
 * this appears to be a problem using the tree regeneration
 * HMM SOMEHOW WE GET nans.. WTF
 * 
 * 
 * 
 * 
 * 
 * 
 * 
 * 
 * 
 * -- TOOD : Make it so that it doesn't re-compute log of prior each go-roun
 * -- FIX NANs in posteriors sorting -- they mess everything up
 * 
 * -- tHESE CONSTANTS SEEM TO HAVE A REALLY BIG INFLUENCE ON WHAT POSTERIORS END UP LOOKING LIKE.
 * -- To run on CPU: http://code.google.com/p/gpuocelot/
 * -- If we want the prior not to depend on the scale, we must build in some logarithmic steps, and treat them equally
 * -- If we want the likelihood not to depend on the scale, we may want a ratio likelihood... but the downside of that is that its not shift-invariant (so if you shift near zero, things go to hell)
 * - Once you add samples, you can resample for a number of steps and output every number of steps
 * - REPLACE &(x[0]) with &x, I think will work!
 * 
 * TODO:
 * 	- Bundle together a hypothesis so its easy to use, and you can get rid of the posterior return
 * 	- Create a local copy of the input and output arrays
 * 	- Can run at a variety of temperatures
 * 	- We can examine the "health" of the chain by seeing how many times we find the "top" one
 * 	-- Huh, probably by particle re-sampling, we can explore the really good regions of the space even better since that moves everyone to one of those regions
 * 
 * 	- The constant stack should be either as large as program (to handle that) or handled better in computing the output
 * 
 */

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>

#include <vector>
using namespace std;



const int N = 5000;  // Hw many chains?
const int BLOCK_SIZE = 1000;
const int N_BLOCKS = N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1);

char* INPUT_FILE = "datasets/polynomial2.txt"; 

const int MAX_PROGRAM_LENGTH = 75; 

const float PRIOR_TEMPERATURE = 1.0; // WARNING: If you make this >=1, our chain, proposing from the prior, may wander into weird long equiv strings
const int MCMC_ITERATIONS = 1000000; //500000;
const float RESAMPLE_P = 0.15; // when we propose, with what probability do we change each program element?

const float LL_SIGMA = 1.0;

const int OUTER_BLOCKS = 1;
const int TOP_RECOPY = 100; // resample flat among these top hypotheseses
const float STACK_EPSILON = 1e-5; // things less than this in absolute value are collapsed to zero, to prevent things like log(1-1)

#include "src/misc.cu"
#include "src/primitives.cu"
#include "src/hypothesis.cu"
#include "src/programs.cu"

__device__ float f_output(float x, hypothesis* h, float* registers, float* stack) {

	// first zero the stack
	for(int i=0;i<2*MAX_PROGRAM_LENGTH;i++) stack[i] = 0.0; 
	
	registers[X]   = x;
	registers[ONE] = 1.0;
	
// 	int garbageflag = 0; // this is set to 1 if ANY subcomputation results in NAN. This way, we don't have CUDA craziness like passings nans to pow 
	                     // if this is 1, we return nan
	
	int top = MAX_PROGRAM_LENGTH; //  We start in the middle of the stack
	for(int p=0;p<MAX_PROGRAM_LENGTH;p++) { // program pointer
		int op = h->program[p];
		
		// update the virtual registers
		registers[ADD]  = stack[top] + stack[top-1];
		registers[SUB]  = stack[top] - stack[top-1];
		registers[MUL]  = stack[top] * stack[top-1];
		registers[DIV]  = stack[top] / stack[top-1];
		registers[NEG]  = -stack[top];
		registers[POW]  = pow(stack[top], stack[top-1]);
		registers[LOG]  = log(stack[top]);
		registers[EXP]  = exp(stack[top]);
		registers[SIN]  = sin(stack[top]);
		registers[ASIN] = asin(stack[top]);
		
		// the *(op<NUM_OPS&&op>=0) here makes us handle accidentally huge numbers, treating them as 0s
		// what elements changes? If we have a constant, we push; a 1-ary and we replace; a 2-ary and we gobble one
		top += stack_change(op)*(op<NUM_OPS&&op>=0); // so 0 args push, 1 args make no change, 2args eat the top
		
		float newtop = registers[op*(op<NUM_OPS&&op>=0)];
		
		// we correct floating point error towards zero to prevent things like log(1-x/x) have a non-inf value
		// It's actually a pretty amazing fact that this thing will exploit this property of floating point numbers!!
// 		stack[top] = float( abs(newtop) > STACK_EPSILON ) * newtop;
		stack[top] = newtop;
		
// 		garbageflag = garbageflag || ( isnan(newtop) );
	}
	
// 	return stack[top]*(garbageflag==1); //!! THIS WILL MAKE 0 FOR ALL NAN OUTPUT
	return stack[top];
}

__device__ float mul(float x, float y) { return x*y; }
__device__ float div(float x, float y) { return x/y; }
__device__ float add(float x, float y) { return x+y; }
__device__ float sub(float x, float y) { return x-y; }

// Kernel that executes on the CUDA device
// initialize_sample here will make us resample if 1; else we use out_hypothesis as-is and propose form that
__global__ void device_run(int N, int DLEN, float* device_INPUT, float* device_OUTPUT, hypothesis* out_hypotheses, int seed, int initialize_sample)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Set up for the RNG
	int rx = idx*seed; // 123456789; // MUST be 32 bit
	int ry = 362436069;
	int rz = 521288629;
	int rw = 88675123;
	
	float stack[2*MAX_PROGRAM_LENGTH]; // Stack is twice as big as program, and populated with 0s so we can eval, e.g., a whole prog of ADDs
	float registers[NUM_OPS];
	
	hypothesis current_;  hypothesis* current = &current_;
	hypothesis proposal_; hypothesis* proposal = &proposal_;
	
	int program_buf_[MAX_PROGRAM_LENGTH]; // needed below as a buffer program
	int* program_buf = &(program_buf_[0]);
	
	void* swaptmp[2]; // used for swapping without arithmetic
		
	if(initialize_sample) { // randomly initialize if we should
		random_closed_expression(current->program,  rx,ry,rz,rw);
		current->llsigma = LL_SIGMA;
	}
	else { // Else copy over
		memcpy((void*)current, out_hypotheses + idx, sizeof(hypothesis));
	}
		
	compute_posterior(DLEN, device_INPUT, device_OUTPUT, current, registers, stack);
		
	// Now main MCMC iterations	
	for(int mcmci=0;mcmci<MCMC_ITERATIONS;mcmci++) {
		
		
		for(int pp=0;pp<=2;pp++) { // for each kind of proposal

			float fb = 0.0;  // the forward-backward probability
			int len = find_program_length(current->program); // TODO: CHECK OFF BY !
			
			if(pp==0) { // full regeneration
				fb = 0.0;
				proposal->llsigma = current->llsigma;
				random_closed_expression(proposal->program, rx, ry, rz, rw);
				fb += compute_program_prior(proposal) - compute_program_prior(current);
			}
			else if(pp==1) {
				float c = random_lnormal(0.0, 1.0, rx,ry,rz,rw);
				fb = llnormalpdf(c, 1.0) - llnormalpdf(proposal->llsigma, 1.0);
				proposal->llsigma = c;
			}
			else if(pp==2) { // normal tree regen
				
				// first find a proposal location:
				int end_ar = (MAX_PROGRAM_LENGTH-1) - random_int(len-1, rx,ry,rz,rw); // the position of what we replace
				int start_ar = find_close_backwards(current->program, end_ar);
				
				// generate a novel tree in program_buf
				int start_x = random_closed_expression(program_buf, rx,ry,rz,rw);
				special_splice(current->program, start_ar, end_ar, program_buf, start_x, proposal->program); // And insert this where we wanted:
				
				float proposed_len = (float) find_program_length(proposal->program);
				float current_len  = (float) find_program_length(current->program);
				
				// forward-back probability
				fb = (-log(current_len)+compute_program_prior(proposal) ) - (-log(proposed_len)+compute_program_prior(current));
			}
			
// 			else if(pp==3) {
// 				
// 				int pos = (MAX_PROGRAM_LENGTH-1) - random_int(len-1, rx,ry,rz,rw);
// 				
// 				// random insert into a program
// 				for(int i=0;i<MAX_PROGRAM_LENGTH;i++) {
// 					proposal->program[i] = current->program[i+1]*(i<pos) + random_int(NUM_OPS,rx,ry,rz,rw)*(i==pos) + current->program[i]*(i>pos);
// 				}
// 				
// 				fb = (-log(float(len)) - log(NUM_OPSf) ) - (-log(float(len+1)) );
// 				
// 			}
// 			else if(pp==4) {
// 				int pos = (MAX_PROGRAM_LENGTH-1) - random_int(len-1, rx,ry,rz,rw);
// 				
// 				// random delete from a program
// 				proposal->program[0] = 0x0;				
// 				for(int i=1;i<MAX_PROGRAM_LENGTH;i++) {
// 					proposal->program[i] = current->program[i-1]*(i<=pos) + current->program[i]*(i>pos);
// 				}
// 				
// 				fb = (-log(float(len)) ) - (-log(float(len-1)) - log(NUM_OPSf) );
// 				
// 			}
			
			
			
// 			else if(pp==7) {
				
// 				// Simple stupid proposal that just mutate at random:	 
// 				for(int i=0;i<MAX_PROGRAM_LENGTH;i++) {
// 					int r = random_float(rx,ry,rz,rw) < RESAMPLE_P;
// 					proposal->program[i] = ifthen(r, random_int(NUM_OPS,rx,ry,rz,rw), current->program[i]);
// 				}
// 			}
			
			// compute the posterior for the proposal
			compute_posterior(DLEN, device_INPUT, device_OUTPUT, proposal, registers, stack);
			
			// compute whether not we accept the proposal, while rejecting infs and nans
			int swap = (random_float(rx,ry,rz,rw) < exp(proposal->posterior - current->posterior - fb) && is_valid(proposal->posterior)) || !is_valid(current->posterior);
			
			// Use a trick to swap pointers without branching 
			ifthenswap(swap, (void**)&current, (void**)&proposal, swaptmp);
			
		} // end for each proposal kind
	}
	
	if (idx<N) {
// 		int tmp[MAX_PROGRAM_LENGTH] = {1,0,0,1,0,0,1,0,7,10,2,6,1,9,5,0,9,1,2,6,6,1,8,3,10,7,5,10,1,1,0,1,1,3,6,4,2,0,9,5,1,10,1,3,3,0,4,10,6,8,4,2,1,6,10,4,11,4,6,1,5,11,1,11,9,9,3,0,0,1,6,10,4,4,3};
// 		memcpy(current->program, tmp, sizeof(tmp));
// 		compute_posterior(DLEN, device_INPUT, device_OUTPUT, current, registers, stack);
// // 		current->posterior = f_output( 2.0, current, registers, stack); //compute_likelihood(DLEN, device_INPUT, device_OUTPUT, current, registers, stack);  
// 		
// 		float x = 2.0;
// 		current->posterior = sub(mul(mul(sin(pow(1.0f,x)),x),sub(exp(exp(asin(1.0f))),asin(div(1.0f,pow(mul(asin(mul(sin(pow(1.0f,add(mul(log(pow(sin(mul(x,sub(sub(1.0f,sin(1.0f)),div(exp(x),add(mul(pow(sub(1.0f,1.0f),x),1.0f),1.0f))))),sin(div(-sin(sub(log(1.0f),pow(pow(add(1.0f,exp(x)),div(exp(1.0f),pow(add(sin(-x),1.0f),x))),x))),1.0f)))),x),x))),1.0f)),0.f),0.f))))),0.f);
// // 		
		memcpy(out_hypotheses + idx, (void*)current, sizeof(hypothesis));
	}
}
 



// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// main routine that executes on the host
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------------------------
int main(void)
{	
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
	// The data:
	FILE* fp = fopen(INPUT_FILE, "r");
	if(fp==NULL) { cerr << "*** ERROR: Cannot open file:\t" << INPUT_FILE <<"\n"; return 1;}
	
	vector<float> inx; float x;
	vector<float> iny; float y;
	char* line = NULL; size_t len=0;
	while( getline(&line, &len, fp) != -1) {
		if (sscanf(line, "%f\t%f\n", &x, &y) == 2) { 
			inx.push_back(x);
			iny.push_back(y);
		}
	}
	
	assert(inx.size() == iny.size());
	const int DLEN = inx.size();
	const size_t DATA_BYTE_LEN = DLEN*sizeof(float);
	
	size_t out_hypotheses_size = N * sizeof(hypothesis);
	hypothesis* host_out_hypotheses = new hypothesis[N]; 
	hypothesis* device_out_hypotheses; hipMalloc((void **) &device_out_hypotheses, out_hypotheses_size); // device allocate
	 
	
	// copy the read input to the GPU
	float* device_INPUT;
	hipMalloc((void **) &device_INPUT, DATA_BYTE_LEN);
	float* host_INPUT = (float*) malloc(DATA_BYTE_LEN);
	for(int i=0;i<DLEN;i++) host_INPUT[i] = inx[i];
	hipMemcpy(device_INPUT, host_INPUT, DATA_BYTE_LEN, hipMemcpyHostToDevice);
	
	float* device_OUTPUT;
	hipMalloc((void **) &device_OUTPUT, DATA_BYTE_LEN);
	float* host_OUTPUT = (float*) malloc(DATA_BYTE_LEN);
	for(int i=0;i<DLEN;i++) host_OUTPUT[i] = iny[i];
	hipMemcpy(device_OUTPUT, host_OUTPUT, DATA_BYTE_LEN, hipMemcpyHostToDevice);
	
	// Do calculation on device:
	srand(time(NULL));
	int seed = rand();
	
	for(int outer=0;outer<OUTER_BLOCKS;outer++) {
		
		device_run<<<N_BLOCKS,BLOCK_SIZE>>>(N, DLEN, device_INPUT, device_OUTPUT, device_out_hypotheses, seed+N*outer, outer==0);
		
		// Retrieve result from device and store it in host array
		hipMemcpy(host_out_hypotheses, device_out_hypotheses, out_hypotheses_size, hipMemcpyDeviceToHost);
		
		// sort the samples by probability:
		qsort( (void*)host_out_hypotheses, N, sizeof(hypothesis), hypothesis_posterior_compare);
		
		//Print results
		for(int n=0; n<N; n++){
			printf("%d\t%d\t%4.2f\t%4.2f\t%4.2f\t%4.2f\t", outer, n, host_out_hypotheses[n].posterior,  host_out_hypotheses[n].prior, host_out_hypotheses[n].likelihood, host_out_hypotheses[n].llsigma);
			
			printf("\"");
			for(int i=0;i<MAX_PROGRAM_LENGTH;i++) 
				printf("%d ", host_out_hypotheses[n].program[i]);
			printf("\"\t");
					
			printf("\"");
			print_program_as_expression( &(host_out_hypotheses[n]) );
			printf("\"\n");
		}
		fflush(stdout);
		
		
// 		TODO: FIX THIS SO THAT IF WE DON'T US ETHIs, It SHOULDN'T FUCK UP THE OHTER RESULTS!!
		
		// Now go through and find the top TOP_RECOPY structurally unique hypotheses
		// and put them at the end
// 		int j = N-1;
// 		for(int i=N;i>=0 && j>=N-TOP_RECOPY;i--) {
// 			int keep = 1;
// 			
// 			for(int chk=j+1;chk<N;chk++){ // check to see if this is identical to anything previous that we've kept
// 				if(hypothesis_structurally_identical(  &(host_out_hypotheses[i]),  &(host_out_hypotheses[chk] )) ) {
// 					keep = 0;
// 					break;
// 				}
// 			}
// 			
// 			if(keep) {
// 				memcpy( (void*)&(host_out_hypotheses[j]), (void*)&(host_out_hypotheses[i]), sizeof(hypothesis)); 
// 				j--; 
// 			}
// 		}
// 		
// //		 We will take the top N and recopy along in the host
// 		for(int i=N-1;i>=N-TOP_RECOPY;i--) { // copy the best, which start from the end
// 			for(int j=i-TOP_RECOPY;j>=0; j -= TOP_RECOPY) // and go backwards
// 				memcpy( (void*)&(host_out_hypotheses[j]), (void*)&(host_out_hypotheses[i]), sizeof(hypothesis)); 
// 			
// 		}
// 		
// 		hipMemcpy(device_out_hypotheses, host_out_hypotheses, out_hypotheses_size, hipMemcpyHostToDevice);
		
	}

	// Cleanup
	delete[] host_out_hypotheses;
	delete[] host_OUTPUT;
	delete[] host_INPUT;
	
	hipFree(device_OUTPUT);
	hipFree(device_INPUT);
	hipFree(device_out_hypotheses);
}